#include <stdio.h>
#include <sys/stat.h>
#include <stdlib.h>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/mman.h>

// FLOAT will either be a float or double depending on what user decides. (could use a better name)
typedef struct {
  int rows;
  int cols;
  FLOAT *arr;          // array of data
  int mmapFileSize;    // size of file mapped to memory
  char *mmapFileLoc;   // pointer to file mapped to memory
} Matrix;

static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/**
* This function will take a filename and map its contents into memory for faster access.
*/
int mapFileToMemory(char* fName, Matrix *mat){
  int fd, size;
  char *map;
  struct stat st;
  
  stat(fName, &st);
  size = st.st_size;      // use stat to get file size
  mat->mmapFileSize = size;
  
  fd = open(fName, O_RDONLY);
  if (fd == -1) {
    perror("Error, could not open file");
    close(fd);
    exit(1);
  }
  
  map = (char*) mmap(0, size, PROT_READ, MAP_PRIVATE, fd, 0);
  if (map == MAP_FAILED) {
    perror("Error, could not map file");
    close(fd);
    exit(1);
  }
  
  mat->mmapFileLoc = map;
  
  close(fd);
  return  0;
}

/**
* This function will un-map a file that was previously mapped into memory.
*/
int unmapFile(Matrix *mat){
  if (munmap(mat->mmapFileLoc, mat->mmapFileSize) == -1) {
  	perror("Error un-mapping the file");
  	exit(1);
  }
  
  return 0;
}

/**
* Write results to a file named "results.out"
*/
void writeOutput(Matrix *mat){
  FILE* ofp;
  ofp = fopen("result.out","w");
  if(ofp == NULL) {
    perror("Could not open result.out to write results");
    exit(1);
  }
  
  int i,j;
  for(i = 0; i < mat->rows; i++){
    for(j = 0; j < mat->cols; j++){
      fprintf(ofp, "%.2f ", mat->arr[i*mat->cols+j]);
    }
    // print newline for all rows
    fprintf(ofp, "\n");
  }
  
  // close output file pointer
  fclose(ofp);
}

/**
* If array holding matrix data is not big enough create a new one twice as big.
* Copy old array data to new array, and free old array from memory.
*/
void doubleArraySize(Matrix *mat) {
  // malloc new array, double the size of previous array
  FLOAT *newArray = (FLOAT *) malloc(sizeof(FLOAT) * mat->rows * mat->cols * 2);
  
  if (newArray == NULL) {
    perror("Error, couldn't allocate space for array\n");
    exit(1);
  }
  
  // copy old array to newArray
  newArray = (FLOAT*) memcpy(newArray, mat->arr, sizeof(FLOAT) * mat->rows * mat->cols);
  
  // free the old array
  free(mat->arr);
  
  // set pointer to new array in memory
  mat->arr = newArray;
}

int main( int argc, char **argv ) {
  int c;
  int *dev_c;
  HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

  //mult<<<1,1>>>( 2, 7, dev_c );

  HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
        hipMemcpyDeviceToHost ) );
  printf( "2 * 7 = %d\n", c );
  HANDLE_ERROR( hipFree( dev_c ) );

  return 0;
}

